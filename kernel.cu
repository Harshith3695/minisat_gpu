#include "hip/hip_runtime.h"
#include <iostream>
#define N 1024

namespace GPU_KERNEL{
using namespace std;

	__shared__ int UE_list[N];

	__shared__ int UE_threads[N];

	__device__
	int var_f(int blocker){
		int var_tmp = blocker >> 1;
		return var_tmp;
	}

	__device__
	int sign_f(int blkr){
		int sign_tmp = blkr & 1;
		return sign_tmp;
	}

	__global__
	void propagate_kernel(int *dev_Lit, int *dev_N, int *dev_blockers, int *dev_flags, int *dev_clauses, int *dev_cindex, int *dev_assigns)
	{

		int tid = threadIdx.x;
		if(tid < *dev_N){

			if(tid == 0){
				int p, q;
				for(p=0;p<=(*dev_N);p++){
					UE_list[p] = -2;
					UE_threads[p] = -2;
					if(p == (*dev_N)){
						UE_list[p] = -1;
						UE_threads[p] = -1;
					}
				}
			}

			int value_1 = (dev_assigns[var_f(dev_blockers[tid])]) ^ (sign_f(dev_blockers[tid]));
			if (value_1 == 0){
				dev_flags[tid] = dev_flags[tid] | 32;
				goto kernel_end;}

			int false_lit = *dev_Lit ^ 1;
			if(dev_clauses[dev_cindex[tid]] == false_lit){
				dev_clauses[dev_cindex[tid]] = dev_clauses[dev_cindex[tid] + 1], dev_clauses[dev_cindex[tid] + 1] = false_lit;
				dev_flags[tid] = dev_flags[tid] | 16;}

			int first = dev_clauses[dev_cindex[tid]];

			UE_threads[tid] = tid, UE_list[tid] = first;

//			int cnt = 0;
//			for(int x=0; UE_list[x] != -1; x++){
//				for(int y=x+1; UE_list[y] != -1; y++){
//					while(UE_list[x] != -2){
//						if(UE_list[x] == UE_list[y])
//							cnt++;
//					}
//				}
//			}


			int value_2 = dev_assigns[var_f(first)] ^ sign_f(first);
			if(first != dev_blockers[tid] && value_2 == 0){
				dev_flags[tid] = dev_flags[tid] | 8;
				goto kernel_end;}

			for(int k = 2; dev_clauses[dev_cindex[tid]+k] != -2 ; k++){
				int value_3 = dev_assigns[var_f(dev_clauses[dev_cindex[tid]+k])] ^ sign_f(dev_clauses[dev_cindex[tid]+k]);
				if(value_3 != 1){
					dev_clauses[dev_cindex[tid]+1] = dev_clauses[dev_cindex[tid]+k]; dev_clauses[dev_cindex[tid]+k] = false_lit;
					dev_flags[tid] = dev_flags[tid] | 4;
					goto kernel_end;
				}
			}

			int value_4 = dev_assigns[var_f(first)] ^ sign_f(first);
			if(value_4 == 1){
				dev_flags[tid] = dev_flags[tid] | 2;
			}else{
	 			dev_assigns[var_f(first)] = (sign_f(first));
				dev_flags[tid] = dev_flags[tid] | 1;

				for(int a=0;a<=(*dev_N);a++){
					if(((UE_threads[a] != -2) && (UE_threads[a] != -1)) && (UE_threads[a] == tid)){
						for(int b=0;UE_list[b] != -1;b++){
							printf("UE_list[%d] = %d, Address = %p, tid = %d\n", a, UE_list[b], &UE_list[b], tid);
						}
					}
					break;
				}
			}

			kernel_end:
			__syncthreads();
		}
	}
};



//			if(tid == 0){
//			for(int a=0;a<=(*dev_N);a++){
//				printf("UE_list[%d] = %d, Address = %p\n", a, UE_list[a], &UE_list[a]);
//				printf("UE_threads[%d] = %d, Address = %p\n", a, UE_threads[a], &UE_threads[a]);
//				}
//			}


//		printf("Kernel = %d\n", tid);
//		printf("dev_Lit = %d, dev_N = %d, dev_blockers = %d,  dev_flags = %d, dev_clauses = %d, dev_cindex = %d, assigns = %d\n", *dev_Lit, *dev_N, dev_blockers[tid], dev_flags[tid], dev_clauses[tid], dev_cindex[tid], dev_assigns[tid]);

//		printf("Clauses = ");
//		for(int a = 0; dev_clauses[dev_cindex[tid]+a] != -2 ; a++)
//			printf("%d, ", dev_clauses[dev_cindex[tid]+a]);
//		printf("\n");

//				printf("flag = %d", dev_flags[tid]);

//			int cnt = 0;
//			while(dev_clauses[dev_cindex[tid]+cnt] != -2){
//				cnt++;
//			}

//			printf("cnt = %d\n", cnt);
//					printf("After: c[1] = %d, c[k] = %d\n", dev_clauses[dev_cindex[tid]+1], dev_clauses[dev_cindex[tid]+k]);

//				printf("Here!");

//			if( value_4 == false_lit){
//				printf("assertion true proceed!");
//			}
//			printf("tid = %d, Kernel_blocker = %d, value_1 = %d\n", tid, dev_blockers[tid], value_1);


// 			atomicAdd(&(dev_assigns[var_f(first)]), sign_f(first));