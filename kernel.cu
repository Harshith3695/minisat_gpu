#include "hip/hip_runtime.h"
#include <iostream>

namespace GPU_KERNEL{
using namespace std;

	__device__
	int var_f(int blocker){
		int var_tmp = blocker >> 1;
		return var_tmp;
	}

	__device__
	int sign_f(int blkr){
		int sign_tmp = blkr & 1;
		return sign_tmp;
	}

	__global__
	void propagate_kernel(int *dev_Lit, int *dev_N, int *dev_blockers, int *dev_flags, int *dev_clauses, int *dev_cindex, int *dev_assigns)
	{

		int tid = blockIdx.x;
		if(tid < *dev_N){

			int value_1 = (dev_assigns[var_f(dev_blockers[tid])]) ^ (sign_f(dev_blockers[tid]));
			if (value_1 == 0){
				dev_flags[tid] = dev_flags[tid] | 32;
				goto kernel_end;}

			int false_lit = *dev_Lit ^ 1;
			if(dev_clauses[dev_cindex[tid]] == false_lit){
				dev_clauses[dev_cindex[tid]] = dev_clauses[dev_cindex[tid] + 1], dev_clauses[dev_cindex[tid] + 1] = false_lit;
				dev_flags[tid] = dev_flags[tid] | 16;}

			int first = dev_clauses[dev_cindex[tid]];
			int value_2 = dev_assigns[var_f(first)] ^ sign_f(first);
			if(first != dev_blockers[tid] && value_2 == 0){
				dev_flags[tid] = dev_flags[tid] | 8;
				goto kernel_end;}

			for(int k = 2; dev_clauses[dev_cindex[tid]+k] != -2 ; k++){
				int value_3 = dev_assigns[var_f(dev_clauses[dev_cindex[tid]+k])] ^ sign_f(dev_clauses[dev_cindex[tid]+k]);
				if(value_3 != 1){
					dev_clauses[dev_cindex[tid]+1] = dev_clauses[dev_cindex[tid]+k]; dev_clauses[dev_cindex[tid]+k] = false_lit;
					dev_flags[tid] = dev_flags[tid] | 4;
					goto kernel_end;
				}
			}

			int value_4 = dev_assigns[var_f(first)] ^ sign_f(first);
//			printf("value_4 = %d, Kernel = %d\n", value_4, tid);
			if(value_4 == 1){
				dev_flags[tid] = dev_flags[tid] | 2;
			}else{
				atomicAdd(&(dev_assigns[var_f(first)]), sign_f(first));
//			printf("tp = %d, Kernel = %d\n", tp, tid);
				dev_flags[tid] = dev_flags[tid] | 1;
			}

			kernel_end:
			__syncthreads();
		}
	}

};


//		printf("Kernel = %d\n", tid);
//		printf("dev_Lit = %d, dev_N = %d, dev_blockers = %d,  dev_flags = %d, dev_clauses = %d, dev_cindex = %d, assigns = %d\n", *dev_Lit, *dev_N, dev_blockers[tid], dev_flags[tid], dev_clauses[tid], dev_cindex[tid], dev_assigns[tid]);

//		printf("Clauses = ");
//		for(int a = 0; dev_clauses[dev_cindex[tid]+a] != -2 ; a++)
//			printf("%d, ", dev_clauses[dev_cindex[tid]+a]);
//		printf("\n");

//				printf("flag = %d", dev_flags[tid]);

//			int cnt = 0;
//			while(dev_clauses[dev_cindex[tid]+cnt] != -2){
//				cnt++;
//			}

//			printf("cnt = %d\n", cnt);
//					printf("After: c[1] = %d, c[k] = %d\n", dev_clauses[dev_cindex[tid]+1], dev_clauses[dev_cindex[tid]+k]);

//				printf("Here!");

//			if( value_4 == false_lit){
//				printf("assertion true proceed!");
//			}
//			printf("tid = %d, Kernel_blocker = %d, value_1 = %d\n", tid, dev_blockers[tid], value_1);
